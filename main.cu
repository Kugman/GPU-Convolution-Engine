
#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <chrono>

#define N 200
#define K 3

using namespace std;
using Matrix = vector<vector<float>>;

Matrix conv2D_CPU(const Matrix& input, const Matrix& kernel) {
   
    int n = input.size(),
        k = kernel.size(),
        pad = k / 2;

    Matrix output(n, vector<float>(n, 0));

    for (int i = pad; i < n - pad; i++) {
        for (int j = pad; j < n - pad; j++) {
            float sum = 0.0f;
            for (int ki = -pad; ki <= pad; ki++) {
                for (int kj = -pad; kj <= pad; kj++) {
                    sum += input[i + ki][j + kj] * kernel[ki + pad][kj + pad];
                }
            }
            output[i][j] = sum;
        }
    }
    return output;
}

__global__ void conv2D_GPU(const float *__restrict__ input, const float * __restrict__ kernel, float *__restrict__ output, const int n, const int k) {
    
    extern __shared__ float shmem[];
    float* tile = shmem;
    float* shKernel = (float*)&tile[(blockDim.y + k - 1) * (blockDim.x + k - 1)];
    
    int pad = k / 2;
    int tile_size_x = blockDim.x;
    int tile_size_y = blockDim.y;

    int row = blockIdx.y * tile_size_y + threadIdx.y;
    int col = blockIdx.x * tile_size_x + threadIdx.x;

    int shared_row = threadIdx.y + pad;
    int shared_col = threadIdx.x + pad;  
    
    if (threadIdx.y < k && threadIdx.x < k)
        shKernel[threadIdx.y * k + threadIdx.x] = kernel[threadIdx.y * k + threadIdx.x];
    
    int global_row = row - pad;
    int global_col = col - pad;
    if (shared_row < tile_size_y + 2 * pad && shared_col < tile_size_x + 2 * pad) {
        if (global_row >= 0 && global_row < n && global_col >= 0 && global_col < n)
            tile[shared_row * (tile_size_x + 2 * pad) + shared_col] = input[global_row * n + global_col];
        else
            tile[shared_row * (tile_size_x + 2 * pad) + shared_col] = 0.0f;
    }

    __syncthreads();

    if (row < pad || row >= n - pad || col < pad || col >= n - pad) return;

    float sum = 0.0f;
    
    for (int i = -pad; i <= pad; i++) 
        for (int j = -pad; j <= pad; j++)
            sum += tile[(shared_row + i) * (tile_size_x + 2 * pad) + (shared_col + j)] * shKernel[(i + pad) * k + (j + pad)];

    output[row * n + col] = sum;
    
}

void cpyToHost(const Matrix &m, float* h, const int size) {
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            h[i * size + j] = m[i][j];    
}

int main()
{
    Matrix input(N, vector<float>(N));
    Matrix kernel = {
        {1, 1, 1},
        {1, 1, 1},
        {1, 1, 1}
    };
    
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            input[i][j] = i % 255;
    
    auto start_cpu = chrono::high_resolution_clock::now();
    Matrix cpu_output = conv2D_CPU(input, kernel);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> cpu_time = end_cpu - start_cpu;

    float h_input[N * N], h_kernel[K*K], h_output[N * N] = {0};
    cpyToHost(input, h_input, N);
    cpyToHost(kernel, h_kernel, K);


    float* d_input, * d_kernel, * d_output;

    hipMalloc((void**)&d_input, N * N * sizeof(float));
    hipMalloc((void**)&d_kernel, K * K * sizeof(float));
    hipMalloc((void**)&d_output, N * N * sizeof(float));

    hipMemcpy(d_input, h_input, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, K * K * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    int sharedMemSize = (threadsPerBlock.x + K - 1) * (threadsPerBlock.y + K - 1) * sizeof(float) + K * K * sizeof(float);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    conv2D_GPU <<<numBlocks, threadsPerBlock, sharedMemSize>>> (d_input, d_kernel, d_output, N, K);
    hipEventRecord(stop);

    hipMemcpy(h_output, d_output, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    cout << "GPU Time: " << gpu_time << " ms" << endl;
    //cout << "Convolution output : " << endl;
    //for (int i = 0; i < N; i++) {
    //    for (int j = 0; j < N; j++)
    //        cout << h_output[i * N + j] << "\t";
    //    cout << endl;
    //}
    //cout << endl;

    
    cout << "CPU Time: " << cpu_time.count() << " ms" << endl;
    //cout << "Convolution output : " << endl;
    //for (auto& row : cpu_output) {
    //    for (auto& val : row)
    //        cout << val << "\t";
    //    cout << endl;
    //}


    return 0;
}
